#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
This benchmark runs distributed join on random keys. Both the left and the right tables contain two
columns. The key column consists of random integers and the payload column consists of row ids.

Parameters:

**--key-type {int32_t,int64_t}**

Data type for the key columns. Default: `int64_t`.

**--payload-type {int32_t,int64_t}**

Data type for the payload columns. Default: `int64_t`.

**--build-table-nrows [INTEGER]**

Number of rows in the build table per GPU. Default: `100'000'000`.

**--probe-table-nrows [INTEGER]**

Number of rows in the probe table per GPU. Default: `100'000'000`.

**--selectivity [FLOAT]**

The probability (in range 0.0 - 1.0) of each probe table row has matches in the build table.
Default: `0.3`.

**--duplicate-build-keys**

If specified, key columns of the build table are allowed to have duplicates.

**--over-decomposition-factor [INTEGER]**

Partition the input tables into (over decomposition factor) * (number of GPUs) buckets, which is
used for computation-communication overlap. This argument has to be an integer >= 1. Higher number
means smaller batch size. `1` means no overlap. Default: `1`.

**--communicator [STR]**

This option can be either "UCX" or "NCCL", which controls what communicator to use. Default: `UCX`.

**--registration-method [STR]**

If the UCX communicator is selected, this option can be either "none", "preregistered" or "buffer",
to control how registration is performed for GPUDirect RDMA.
- "none": No preregistration.
- "preregistered": The whole RMM memory pool will be preregistered.
- "buffer": Preregister a set of communication buffers. The communication in distributed join will
go through these buffers.
*/

#include "../src/communicator.hpp"
#include "../src/compression.hpp"
#include "../src/distributed_join.hpp"
#include "../src/error.hpp"
#include "../src/generate_table.cuh"
#include "../src/registered_memory_resource.hpp"
#include "../src/setup.hpp"

#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include <hip/hip_runtime_api.h>

#include <mpi.h>

#include <algorithm>
#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <string>
#include <tuple>
#include <utility>
#include <vector>

static std::string key_type     = "int64_t";
static std::string payload_type = "int64_t";

static cudf::size_type BUILD_TABLE_NROWS_EACH_RANK = 100'000'000;
static cudf::size_type PROBE_TABLE_NROWS_EACH_RANK = 100'000'000;
static double SELECTIVITY                          = 0.3;
static bool IS_BUILD_TABLE_KEY_UNIQUE              = true;
static int OVER_DECOMPOSITION_FACTOR               = 1;
static std::string COMMUNICATOR_NAME               = "UCX";
static std::string REGISTRATION_METHOD             = "preregistered";
static int64_t COMMUNICATOR_BUFFER_SIZE            = 1'600'000'000LL;
static bool COMPRESSION                            = false;
static int NVLINK_DOMAIN_SIZE                      = 1;
static bool REPORT_TIMING                          = false;

void parse_command_line_arguments(int argc, char *argv[])
{
  for (int iarg = 0; iarg < argc; iarg++) {
    if (!strcmp(argv[iarg], "--key-type")) { key_type = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--payload-type")) { payload_type = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--build-table-nrows")) {
      BUILD_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--probe-table-nrows")) {
      PROBE_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--selectivity")) { SELECTIVITY = atof(argv[iarg + 1]); }

    if (!strcmp(argv[iarg], "--duplicate-build-keys")) { IS_BUILD_TABLE_KEY_UNIQUE = false; }

    if (!strcmp(argv[iarg], "--over-decomposition-factor")) {
      OVER_DECOMPOSITION_FACTOR = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--communicator")) { COMMUNICATOR_NAME = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--compression")) { COMPRESSION = true; }

    if (!strcmp(argv[iarg], "--registration-method")) { REGISTRATION_METHOD = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--nvlink-domain-size")) { NVLINK_DOMAIN_SIZE = atoi(argv[iarg + 1]); }

    if (!strcmp(argv[iarg], "--report-timing")) { REPORT_TIMING = true; }
  }
}

void report_configuration()
{
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

  int mpi_rank;
  int mpi_size;
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));
  if (mpi_rank != 0) return;

  std::cout << "========== Parameters ==========" << std::endl;
  std::cout << std::boolalpha;
  std::cout << "Key type: " << key_type << std::endl;
  std::cout << "Payload type: " << payload_type << std::endl;
  std::cout << "Number of rows in the build table: "
            << static_cast<uint64_t>(BUILD_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6 << " million"
            << std::endl;
  std::cout << "Number of rows in the probe table: "
            << static_cast<uint64_t>(PROBE_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6 << " million"
            << std::endl;
  std::cout << "Selectivity: " << SELECTIVITY << std::endl;
  std::cout << "Keys in build table are unique: " << IS_BUILD_TABLE_KEY_UNIQUE << std::endl;
  std::cout << "Over-decomposition factor: " << OVER_DECOMPOSITION_FACTOR << std::endl;
  std::cout << "Communicator: " << COMMUNICATOR_NAME << std::endl;
  if (COMMUNICATOR_NAME == "UCX")
    std::cout << "Registration method: " << REGISTRATION_METHOD << std::endl;
  std::cout << "Compression: " << COMPRESSION << std::endl;
  std::cout << "NVLink domain size: " << NVLINK_DOMAIN_SIZE << std::endl;
  std::cout << "================================" << std::endl;
}

int main(int argc, char *argv[])
{
  MPI_CALL(MPI_Init(&argc, &argv));
  set_cuda_device();

  /* Parse command line arguments */

  parse_command_line_arguments(argc, argv);
  report_configuration();

  cudf::size_type RAND_MAX_VAL =
    std::max(BUILD_TABLE_NROWS_EACH_RANK, PROBE_TABLE_NROWS_EACH_RANK) * 2;

  /* Initialize communicator and memory pool */

  int mpi_rank;
  int mpi_size;
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));

  Communicator *communicator{nullptr};
  // `registered_mr` holds reference to the registered memory resource, and *nullptr* if registered
  // memory resource is not used.
  registered_memory_resource *registered_mr{nullptr};
  // pool_mr need to live on heap because for registered memory resources, the memory pool needs
  // to deallocated before UCX cleanup, which can be achieved by calling the destructor of
  // `poll_mr`.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> *pool_mr{nullptr};

  setup_memory_pool_and_communicator(communicator,
                                     registered_mr,
                                     pool_mr,
                                     COMMUNICATOR_NAME,
                                     REGISTRATION_METHOD,
                                     COMMUNICATOR_BUFFER_SIZE);

  void *preallocated_pinned_buffer;
  CUDA_RT_CALL(hipHostMalloc(&preallocated_pinned_buffer, mpi_size * sizeof(size_t)));

  /* Warmup nvcomp */

  if (COMPRESSION) { warmup_nvcomp(); }

  /* Generate build table and probe table on each rank */

  std::unique_ptr<cudf::table> left;
  std::unique_ptr<cudf::table> right;

#define generate_tables(KEY_T, PAYLOAD_T)                                        \
  {                                                                              \
    std::tie(left, right) =                                                      \
      generate_tables_distributed<KEY_T, PAYLOAD_T>(BUILD_TABLE_NROWS_EACH_RANK, \
                                                    PROBE_TABLE_NROWS_EACH_RANK, \
                                                    SELECTIVITY,                 \
                                                    RAND_MAX_VAL,                \
                                                    IS_BUILD_TABLE_KEY_UNIQUE,   \
                                                    communicator);               \
  }

#define generate_tables_key_type(KEY_T)                 \
  {                                                     \
    if (payload_type == "int64_t") {                    \
      generate_tables(KEY_T, int64_t)                   \
    } else if (payload_type == "int32_t") {             \
      generate_tables(KEY_T, int32_t)                   \
    } else {                                            \
      throw std::runtime_error("Unknown payload type"); \
    }                                                   \
  }

  if (key_type == "int64_t") {
    generate_tables_key_type(int64_t)
  } else if (key_type == "int32_t") {
    generate_tables_key_type(int32_t)
  } else {
    throw std::runtime_error("Unknown key type");
  }

  /* Generate compression options */

  std::vector<ColumnCompressionOptions> left_compression_options =
    generate_compression_options_distributed(left->view(), COMPRESSION);
  std::vector<ColumnCompressionOptions> right_compression_options =
    generate_compression_options_distributed(right->view(), COMPRESSION);

  /* Distributed join */

  CUDA_RT_CALL(hipDeviceSynchronize());

  MPI_Barrier(MPI_COMM_WORLD);
  hipProfilerStart();
  double start = MPI_Wtime();

  std::unique_ptr<cudf::table> join_result = distributed_inner_join(left->view(),
                                                                    right->view(),
                                                                    {0},
                                                                    {0},
                                                                    communicator,
                                                                    left_compression_options,
                                                                    right_compression_options,
                                                                    OVER_DECOMPOSITION_FACTOR,
                                                                    REPORT_TIMING,
                                                                    preallocated_pinned_buffer,
                                                                    NVLINK_DOMAIN_SIZE);

  MPI_Barrier(MPI_COMM_WORLD);
  double stop = MPI_Wtime();
  hipProfilerStop();

  if (mpi_rank == 0) { std::cout << "Elasped time (s) " << stop - start << std::endl; }

  /* Cleanup */
  left.reset();
  right.reset();
  join_result.reset();
  CUDA_RT_CALL(hipHostFree(preallocated_pinned_buffer));
  CUDA_RT_CALL(hipDeviceSynchronize());

  destroy_memory_pool_and_communicator(
    communicator, registered_mr, pool_mr, COMMUNICATOR_NAME, REGISTRATION_METHOD);

  MPI_CALL(MPI_Finalize());

  return 0;
}
