#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <string>
#include <tuple>
#include <utility>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <mpi.h>

#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include "../src/communicator.h"
#include "../src/distributed_join.cuh"
#include "../src/error.cuh"
#include "../src/generate_table.cuh"
#include "../src/registered_memory_resource.hpp"
#include "../src/topology.cuh"

static std::string key_type     = "int64_t";
static std::string payload_type = "int64_t";

static cudf::size_type BUILD_TABLE_NROWS_EACH_RANK = 100'000'000;
static cudf::size_type PROBE_TABLE_NROWS_EACH_RANK = 100'000'000;
static double SELECTIVITY                          = 0.3;
static bool IS_BUILD_TABLE_KEY_UNIQUE              = true;
static int OVER_DECOMPOSITION_FACTOR               = 1;
static std::string COMMUNICATOR_NAME               = "UCX";
static bool USE_BUFFER_COMMUNICATOR                = false;
static int64_t COMMUNICATOR_BUFFER_SIZE            = 1'600'000'000LL;

void parse_command_line_arguments(int argc, char *argv[])
{
  for (int iarg = 0; iarg < argc; iarg++) {
    if (!strcmp(argv[iarg], "--key-type")) { key_type = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--payload-type")) { payload_type = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--build-table-nrows")) {
      BUILD_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--probe-table-nrows")) {
      PROBE_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--selectivity")) { SELECTIVITY = atof(argv[iarg + 1]); }

    if (!strcmp(argv[iarg], "--duplicate-build-keys")) { IS_BUILD_TABLE_KEY_UNIQUE = false; }

    if (!strcmp(argv[iarg], "--over-decomposition-factor")) {
      OVER_DECOMPOSITION_FACTOR = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--communicator")) { COMMUNICATOR_NAME = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--use-buffer-communicator")) { USE_BUFFER_COMMUNICATOR = true; }
  }
}

void report_configuration()
{
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

  int mpi_rank;
  int mpi_size;
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));
  if (mpi_rank != 0) return;

  std::cout << "========== Parameters ==========" << std::endl;
  std::cout << std::boolalpha;
  std::cout << "Key type: " << key_type << std::endl;
  std::cout << "Payload type: " << payload_type << std::endl;
  std::cout << "Number of rows in the build table: "
            << static_cast<uint64_t>(BUILD_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6 << " million"
            << std::endl;
  std::cout << "Number of rows in the probe table: "
            << static_cast<uint64_t>(PROBE_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6 << " million"
            << std::endl;
  std::cout << "Selectivity: " << SELECTIVITY << std::endl;
  std::cout << "Keys in build table are unique: " << IS_BUILD_TABLE_KEY_UNIQUE << std::endl;
  std::cout << "Over-decomposition factor: " << OVER_DECOMPOSITION_FACTOR << std::endl;
  std::cout << "Communicator: " << COMMUNICATOR_NAME << std::endl;
  if (COMMUNICATOR_NAME == "UCX")
    std::cout << "Buffer communicator: " << USE_BUFFER_COMMUNICATOR << std::endl;
  std::cout << "================================" << std::endl;
}

int main(int argc, char *argv[])
{
  /* Initialize topology */

  setup_topology(argc, argv);

  /* Parse command line arguments */

  parse_command_line_arguments(argc, argv);
  report_configuration();

  cudf::size_type RAND_MAX_VAL =
    std::max(BUILD_TABLE_NROWS_EACH_RANK, PROBE_TABLE_NROWS_EACH_RANK) * 2;

  /* Initialize communicator and memory pool */

  int mpi_rank;
  int mpi_size;
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));

  Communicator *communicator{nullptr};
  // `mr` holds reference to the registered memory resource, and *nullptr* if registered memory
  // resource is not used.
  registered_memory_resource *mr{nullptr};
  // pool_mr need to live on heap because for registered memory resources, the memory pool needs
  // to deallocated before UCX cleanup, which can be achieved by calling the destructor of
  // `poll_mr`.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> *pool_mr{nullptr};

  // Calculate the memory pool size
  size_t free_memory, total_memory;
  CUDA_RT_CALL(hipMemGetInfo(&free_memory, &total_memory));
  const size_t pool_size = free_memory - 5LL * (1LL << 29);  // free memory - 500MB

  if (COMMUNICATOR_NAME == "UCX" && USE_BUFFER_COMMUNICATOR) {
    // For UCX with buffer communicator, a memory pool is first constructed so that the
    // communication buffers will be allocated in memory pool.
    pool_mr = new rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource>(
      rmm::mr::get_current_device_resource(), pool_size, pool_size);
    rmm::mr::set_current_device_resource(pool_mr);
    // *2 because buffers are needed for both sends and receives
    const int num_comm_buffers = 2 * mpi_size;
    communicator               = initialize_ucx_communicator(
      true, num_comm_buffers, COMMUNICATOR_BUFFER_SIZE / num_comm_buffers - 100'000LL);
  } else if (COMMUNICATOR_NAME == "UCX" && !USE_BUFFER_COMMUNICATOR) {
    // For UCX with preregistered memory pool, a communicator is first constructed so that
    // `registered_memory_resource` can use the communicator for buffer registrations.
    UCXCommunicator *ucx_communicator = initialize_ucx_communicator(false, 0, 0);
    communicator                      = ucx_communicator;
    mr                                = new registered_memory_resource(ucx_communicator);
    pool_mr =
      new rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource>(mr, pool_size, pool_size);
    rmm::mr::set_current_device_resource(pool_mr);
  } else if (COMMUNICATOR_NAME == "NCCL") {
    communicator = new NCCLCommunicator;
    communicator->initialize();
    pool_mr = new rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource>(
      rmm::mr::get_current_device_resource(), pool_size, pool_size);
    rmm::mr::set_current_device_resource(pool_mr);
  } else {
    throw std::runtime_error("Unknown communicator name");
  }

  /* Generate build table and probe table on each rank */

  std::unique_ptr<cudf::table> left;
  std::unique_ptr<cudf::table> right;

#define generate_tables(KEY_T, PAYLOAD_T)                                        \
  {                                                                              \
    std::tie(left, right) =                                                      \
      generate_tables_distributed<KEY_T, PAYLOAD_T>(BUILD_TABLE_NROWS_EACH_RANK, \
                                                    PROBE_TABLE_NROWS_EACH_RANK, \
                                                    SELECTIVITY,                 \
                                                    RAND_MAX_VAL,                \
                                                    IS_BUILD_TABLE_KEY_UNIQUE,   \
                                                    communicator);               \
  }

#define generate_tables_key_type(KEY_T)                 \
  {                                                     \
    if (payload_type == "int64_t") {                    \
      generate_tables(KEY_T, int64_t)                   \
    } else if (payload_type == "int32_t") {             \
      generate_tables(KEY_T, int32_t)                   \
    } else {                                            \
      throw std::runtime_error("Unknown payload type"); \
    }                                                   \
  }

  if (key_type == "int64_t") {
    generate_tables_key_type(int64_t)
  } else if (key_type == "int32_t") {
    generate_tables_key_type(int32_t)
  } else {
    throw std::runtime_error("Unknown key type");
  }

  /* Distributed join */

  CUDA_RT_CALL(hipDeviceSynchronize());

  MPI_Barrier(MPI_COMM_WORLD);
  hipProfilerStart();
  double start = MPI_Wtime();

  std::unique_ptr<cudf::table> join_result =
    distributed_inner_join(left->view(),
                           right->view(),
                           {0},
                           {0},
                           {std::pair<cudf::size_type, cudf::size_type>(0, 0)},
                           communicator,
                           OVER_DECOMPOSITION_FACTOR);

  MPI_Barrier(MPI_COMM_WORLD);
  double stop = MPI_Wtime();
  hipProfilerStop();

  if (mpi_rank == 0) { std::cout << "Elasped time (s) " << stop - start << std::endl; }

  /* Cleanup */
  left.reset();
  right.reset();
  join_result.reset();
  CUDA_RT_CALL(hipDeviceSynchronize());

  if (USE_BUFFER_COMMUNICATOR) {
    // When finalizing buffer communicator, communication buffers need be deallocated, so
    // `finalize` needs to be called before the memory pool is deleted.
    communicator->finalize();
    delete pool_mr;
    delete mr;
  } else {
    // For registered memory resouce, the memory pool needs to be deleted before finalizing
    // the communicator, so that all buffers can be deregistered through UCX.
    delete pool_mr;
    delete mr;
    communicator->finalize();
  }

  delete communicator;

  MPI_CALL(MPI_Finalize());

  return 0;
}
